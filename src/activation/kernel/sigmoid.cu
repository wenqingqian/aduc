#include "hip/hip_runtime.h"
#include "float4.cuh"
#include <assert.h>

namespace {
	
	__global__
	void kernel(float* x, float* y, int N){
		int tid = blockIdx.x * blockDim.x + threadIdx.x;

		if(tid >= N / 4) return;
		f4 x4 = reinterpret_cast<f4*>(x)[tid];

		#pragma unroll
		for(int i = 0; i < 4; i ++){
			x4[i] = 1.0f / (1.0f + expf(- x4[i]));
		}

		f4* out = reinterpret_cast<f4*>(y);
		out[tid] = x4;
	}
}

void sigmoid(float* x, float* y, int N){
	assert(N % 4 == 0);
	kernel<<<(N + 127) / 128, 32>>>(x, y, N);
}

