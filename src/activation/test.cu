#include <hip/hip_runtime.h>
#include "matrix.h"

extern void sigmoid(float*,float*,int);
extern void relu(float*,float*,int);


int main(){
	printf("sigmoid----------------------------------\n");
	{
		const int N = 8;
		matrix<float> ma(N,1);
		float* x;
		hipMalloc(&x, sizeof(float) * N);
		hipMemcpy(x, ma.unsafe_data(), sizeof(float) * N, hipMemcpyHostToDevice);
		ma.show();
		sigmoid(x, x, N);
		hipMemcpy(ma.unsafe_data(), x, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		ma.show();
	}
	printf("relu----------------------------------\n");
	{
		const int N = 8;
		matrix<float> ma(N,1);
		float* x;
		hipMalloc(&x, sizeof(float) * N);
		hipMemcpy(x, ma.unsafe_data(), sizeof(float) * N, hipMemcpyHostToDevice);
		ma.show();
		relu(x, x, N);
		hipMemcpy(ma.unsafe_data(), x, sizeof(float) * N, hipMemcpyDeviceToHost);
		hipDeviceSynchronize();
		ma.show();
	}
}