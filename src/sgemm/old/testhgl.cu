#include <hipblas.h>

#include <hip/hip_runtime.h>

#include "matrix.h"

#include "util.cuh"
int main(){
	int M = 3584, N = 3584, K = 3584;
	float alpha = 1.2, beta = 1.3;
	matrix<float> A(M,K), B(K,N), C(M,N);
	float * A_, * B_, * C_;
	
	hipMalloc(&A_, M * K * sizeof(float));
	hipMemcpy(A_, A.unsafe_data(), M * K * sizeof(float),
				hipMemcpyHostToDevice);
	hipMalloc(&B_, K * N * sizeof(float));
	hipMemcpy(B_, B.unsafe_data(), K * N * sizeof(float),
				hipMemcpyHostToDevice);
	hipMalloc(&C_, M * N * sizeof(float));
	hipMemcpy(C_, C.unsafe_data(), M * N * sizeof(float),
				hipMemcpyHostToDevice);

	hipDeviceSynchronize();

	extern_gemm(gemmHideGmemLatency);
	gemmHideGmemLatency(A_,B_,C_,alpha,beta,M,N,K);

	extern_gemm(r1_HideGmemLatency);
	r1_HideGmemLatency(A_,B_,C_,alpha,beta,M,N,K);
	hipDeviceSynchronize();
}